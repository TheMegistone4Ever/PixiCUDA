#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <crt/math_functions.hpp>
#include <>
#include "cualgo/motion_blur.cuh"

#ifdef __INTELLISENSE__
#include "cualgo/intellisense/intellisense_cuda_intrinsics.hpp"
#endif

#define DEG_TO_RAD 0.01745329252f

// Function to apply motion blur to an image
__global__ void motion_blur_cuda(
	const unsigned char* in_image,
	unsigned char* out_image,
	const float* kernel,
	const int kernel_size,
	const int channels,
	const int height,
	const int width
);

void cuda_motion_blur_image(
	const unsigned char* in_image,
	unsigned char* out_image,
	const float angle_deg,
	const int distance,
	const int height,
	const int width,
	const int channels
) {
	// Calculate the size of the image
	const size_t image_size = height * width * channels * sizeof(unsigned char);
	
	// Create the kernel and fill it with the correct values

	float angle_rad = angle_deg * DEG_TO_RAD;

	int size = distance * 2 + 1; // +1 to include the center pixel
	int grid_kernel_size = size * size;
	size_t kernel_size = grid_kernel_size * sizeof(float);

	float* kernel = new float[grid_kernel_size];

	for (int i = 0; i < size; i++) {
		int x = distance + int(i * cos(angle_rad));
		int y = distance + int(i * sin(angle_rad));
		if (x < 0 || x >= size || y < 0 || y >= size) {
			break;
		}
		kernel[y * size + x] = 1. / (distance + 1);
	}

	// Allocate CUDA variable memory on the device
	unsigned char* device_in_image = nullptr;
	unsigned char* device_out_image = nullptr;

	hipMalloc((void**)&device_in_image, image_size);
	hipMalloc((void**)&device_out_image, image_size);
	hipMemset(device_out_image, NULL, image_size);

	// Copy the host variables to the device (CPU -> GPU)
	hipMemcpy(device_in_image, in_image, image_size, hipMemcpyHostToDevice);

	// Kernel launch
	dim3 grid_image(width, height);
	motion_blur_cuda << <grid_image, 1 >> > (device_in_image, device_out_image, kernel, kernel_size, channels, height, width);

	// Copy the device variables to the host (GPU -> CPU)
	hipMemcpy(out_image, device_out_image, image_size, hipMemcpyDeviceToHost);

	// Free up the memory on the device (GPU)
	hipFree(device_in_image);
	hipFree(device_out_image);
}

__global__ void motion_blur_cuda(
	const unsigned char* in_image,
	unsigned char* out_image,
	const float* kernel,
	const int kernel_size,
	const int channels,
	const int height,
	const int width
) {
	// Get the position of the pixel
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int index = (x + y * gridDim.x) * channels;

	// Apply the motion blur to the pixel
	if (x < width && y < height) {
		for (int channel = 0; channel < channels; channel++) {
			float sum = 0;
			for (int i = 0; i < kernel_size; i++) {
				int x_kernel = i % kernel_size;
				int y_kernel = i / kernel_size;
				int x_image = x + x_kernel - kernel_size / 2;
				int y_image = y + y_kernel - kernel_size / 2;
				if (x_image >= 0 && x_image < width && y_image >= 0 && y_image < height) {
					atomicAdd(&sum, in_image[(x_image + y_image * width) * channels + channel] * kernel[i]);
				}
			}
			out_image[index + channel] = (int)sum;
		}
	}
}

//__global__ void negative_cuda(
//	unsigned char* image,
//	const int channels
//);
//
//void cuda_negative_image(
//	unsigned char* image,
//	const int height,
//	const int width,
//	const int channels
//) {
//	const int image_size = height * width * channels * sizeof(unsigned char);
//
//	unsigned char* device_image = nullptr;
//
//	// Allocate CUDA variable memory on the device
//	hipMalloc((void**)&device_image, image_size);
//
//	// Copy the host variables to the device (CPU -> GPU)
//	hipMemcpy(device_image, image, image_size, hipMemcpyHostToDevice);
//
//	// Kernel launch
//	dim3 grid_image(width, height);
//	negative_cuda << <grid_image, 1 >> > (device_image, channels);
//
//	// Copy the device variables to the host (GPU -> CPU)
//	hipMemcpy(image, device_image, image_size, hipMemcpyDeviceToHost);
//
//	// Free up the memory on the device (GPU)
//	hipFree(device_image);
//}
//
//__global__ void negative_cuda(
//	unsigned char* image,
//	const int channels
//) {
//	int x = blockIdx.x * blockDim.x + threadIdx.x;
//	int y = blockIdx.y * blockDim.y + threadIdx.y;
//	int index = (x + y * gridDim.x) * channels;
//
//	for (int i = 0; i < channels; i++) {
//		image[index + i] = 255 - image[index + i];
//	}
//}
//
//__global__ void histogram_cuda(
//    const unsigned char* image,
//    const int width,
//    const int height,
//    const int channels,
//    int* histogram_r,
//    int* histogram_g,
//    int* histogram_b
//);
//
//void cuda_calculate_histogram(
//    const unsigned char* image,
//    const int height,
//    const int width,
//    const int channels,
//    int* histogram_r,
//    int* histogram_g,
//    int* histogram_b
//) {
//    const int image_size = height * width * channels * sizeof(unsigned char);
//    const int histogram_size = 256 * sizeof(int);
//
//    unsigned char* device_image = nullptr;
//    int* device_histogram_r = nullptr;
//    int* device_histogram_g = nullptr;
//    int* device_histogram_b = nullptr;
//
//    // Allocate CUDA variable memory on the device
//    hipMalloc((void**)&device_image, image_size);
//    hipMalloc((void**)&device_histogram_r, histogram_size);
//    hipMalloc((void**)&device_histogram_g, histogram_size);
//    hipMalloc((void**)&device_histogram_b, histogram_size);
//
//    // Copy the host variables to the device (CPU -> GPU)
//    hipMemcpy(device_image, image, image_size, hipMemcpyHostToDevice);
//    hipMemcpy(device_histogram_r, histogram_r, histogram_size, hipMemcpyHostToDevice);
//    hipMemcpy(device_histogram_g, histogram_g, histogram_size, hipMemcpyHostToDevice);
//    hipMemcpy(device_histogram_b, histogram_b, histogram_size, hipMemcpyHostToDevice);
//
//    // Kernel launch
//    dim3 grid_image(width, height);
//    dim3 block_image(1, 1);
//    histogram_cuda << <grid_image, block_image >> > (device_image, width, height, channels, device_histogram_r, device_histogram_g, device_histogram_b);
//
//    // Copy the device variables to the host (GPU -> CPU)
//    hipMemcpy(histogram_r, device_histogram_r, histogram_size, hipMemcpyDeviceToHost);
//    hipMemcpy(histogram_g, device_histogram_g, histogram_size, hipMemcpyDeviceToHost);
//    hipMemcpy(histogram_b, device_histogram_b, histogram_size, hipMemcpyDeviceToHost);
//
//    // Free up the memory on the device (GPU)
//    hipFree(device_image);
//    hipFree(device_histogram_r);
//    hipFree(device_histogram_g);
//    hipFree(device_histogram_b);
//}
//
//__global__ void histogram_cuda(
//    const unsigned char* image,
//    const int width,
//    const int height,
//    const int channels,
//    int* histogram_r,
//    int* histogram_g,
//    int* histogram_b
//) {
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//    int index = (y * gridDim.x + x) * channels;
//
//    atomicAdd(&histogram_r[image[index + 0]], 1);
//    atomicAdd(&histogram_g[image[index + 1]], 1);
//    atomicAdd(&histogram_b[image[index + 2]], 1);
//}
