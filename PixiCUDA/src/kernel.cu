#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include "cualgo/cuda_histogram.h"

__global__ void histogram_cuda(
	const unsigned char* image,
	const int channels,
	int* histogram_r,
	int* histogram_g,
	int* histogram_b
);

void cuda_calculate_histogram(
	const unsigned char* image,
	const int height,
	const int width,
	const int channels,
	int* histogram_r,
	int* histogram_g,
	int* histogram_b
) {
	unsigned char* device_image = nullptr;
	const int image_size = height * width * channels * sizeof(int);

	const int histogram_size = 256 * sizeof(int);
	int* device_histogram_r = nullptr;
	int* device_histogram_g = nullptr;
	int* device_histogram_b = nullptr;

	// Allocate CUDA variable memory on the device
	hipMalloc((void**)&device_image, image_size);
	hipMalloc((void**)&device_histogram_r, histogram_size);
	hipMalloc((void**)&device_histogram_g, histogram_size);
	hipMalloc((void**)&device_histogram_b, histogram_size);

	// Copy the host variables to the device (CPU -> GPU)
	hipMemcpy(device_image, image, image_size, hipMemcpyHostToDevice);
	hipMemcpy(device_histogram_r, histogram_r, histogram_size, hipMemcpyHostToDevice);
	hipMemcpy(device_histogram_g, histogram_g, histogram_size, hipMemcpyHostToDevice);
	hipMemcpy(device_histogram_b, histogram_b, histogram_size, hipMemcpyHostToDevice);

	// Kernel launch
	dim3 grid_image(width, height);
	dim3 block_dim(1, 1);
	histogram_cuda<<<grid_image, block_dim>>>(device_image, channels, device_histogram_r, device_histogram_g, device_histogram_b);

	// Copy the device variables to the host (GPU -> CPU)
	hipMemcpy(histogram_r, device_histogram_r, histogram_size, hipMemcpyDeviceToHost);
	hipMemcpy(histogram_g, device_histogram_g, histogram_size, hipMemcpyDeviceToHost);
	hipMemcpy(histogram_b, device_histogram_b, histogram_size, hipMemcpyDeviceToHost);

	// Free up the memory on the device (GPU)
	hipFree(device_image);
	hipFree(device_histogram_r);
	hipFree(device_histogram_g);
	hipFree(device_histogram_b);
}

__global__ void histogram_cuda(
	const unsigned char* image,
	const int channels,
	int* histogram_r,
	int* histogram_g,
	int* histogram_b
) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int index = (y * gridDim.x + x) * channels;

	atomicAdd(&histogram_r[image[index + 0]], 1);
	atomicAdd(&histogram_g[image[index + 1]], 1);
	atomicAdd(&histogram_b[image[index + 2]], 1);
}