#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include "cualgo/cuda_histogram.h"

__global__ void histogram_cuda(
	const unsigned char* image,
	int* histogram_grayscale
);

void cuda_calculate_histogram(
	const unsigned char* image,
	const int height,
	const int width,
	const int channels,
	int* histogram_grayscale
) {
	unsigned char* device_image = nullptr;
	int* device_histogram = nullptr;

	const int image_size = height * width * channels * sizeof(int);
	const int histogram_size = 256 * sizeof(int);

	// Allocate CUDA variable memory on the device
	hipMalloc((void**)&device_image, image_size);
	hipMalloc((void**)&device_histogram, histogram_size);

	// Copy the host variables to the device (CPU -> GPU)
	hipMemcpy(device_image, image, image_size, hipMemcpyHostToDevice);
	hipMemcpy(device_histogram, histogram_grayscale, histogram_size, hipMemcpyHostToDevice);

	// Kernel launch
	dim3 grid_image(width, height);
	dim3 block_dim(1, 1);
	histogram_cuda<<<grid_image, block_dim>>>(device_image, device_histogram);

	// Copy the device variables to the host (GPU -> CPU)
	hipMemcpy(histogram_grayscale, device_histogram, histogram_size, hipMemcpyDeviceToHost);

	// Free up the memory on the device (GPU)
	hipFree(device_image);
	hipFree(device_histogram);
}

__global__ void histogram_cuda(
	const unsigned char* image,
	int* histogram_grayscale
) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int index = y * gridDim.x + x;

	atomicAdd(&histogram_grayscale[image[index]], 1);
}