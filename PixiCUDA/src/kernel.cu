#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <crt/math_functions.hpp>
#include <>
#include "cualgo/motion_blur.cuh"

#include <stdio.h>

#ifdef __INTELLISENSE__
#include "cualgo/intellisense/intellisense_cuda_intrinsics.hpp"
#endif // __INTELLISENSE__

constexpr auto BYTE_SIZE = 8;
#define BIT_VECTOR_SIZE (MAX_KERNEL_SIZE * MAX_KERNEL_SIZE / BYTE_SIZE + (MAX_KERNEL_SIZE * MAX_KERNEL_SIZE % BYTE_SIZE != 0))

__constant__ unsigned char device_kernel[BIT_VECTOR_SIZE] = { 0 };

__host__ void setBit(unsigned char* bvec, const int index)
{
    bvec[index / BYTE_SIZE] |= 1 << (index % BYTE_SIZE);
}

__device__ bool testBit(const unsigned char* bvec, const int index)
{
    return bvec[index / BYTE_SIZE] & (1 << (index % BYTE_SIZE));
}

__global__ void motion_blur_cuda(
    const unsigned char* in_image,
    unsigned char* out_image,
    const int kernel_size,
    const int channels,
    const int height,
    const int width,
    const float angle_deg,
    const int distance
);

void cuda_motion_blur_image(
    const unsigned char* in_image,
    unsigned char* out_image,
    const float angle_deg,
    const int distance,
    const int height,
    const int width,
    const int channels
)
{
    // Calculate the size of the image
    const size_t image_size = height * width * channels * sizeof(unsigned char);

    // Create the kernel and fill it with the correct values
    float angle_rad = angle_deg * DEG_TO_RAD;
    int kernel_size = distance * 2 + 1; // +1 to include the center pixel
    unsigned char* host_kernel = new unsigned char[BIT_VECTOR_SIZE] { 0 };

    for (int i = 0; i < kernel_size; i++)
    {
        int x = distance + int(i * cos(angle_rad));
        int y = distance + int(i * sin(angle_rad));
        if (x >= 0 && x < kernel_size && y >= 0 && y < kernel_size)
        {
            setBit(host_kernel, y * kernel_size + x);
        }
    }

    // Copy the host kernel data to the device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(device_kernel), host_kernel, BIT_VECTOR_SIZE);

    // Allocate CUDA variable memory on the device
    unsigned char* device_in_image = nullptr;
    unsigned char* device_out_image = nullptr;

    hipMalloc((void**)&device_in_image, image_size);
    hipMalloc((void**)&device_out_image, image_size);
    hipMemset(device_out_image, NULL, image_size);

    // Copy the host variables to the device (CPU -> GPU)
    hipMemcpy(device_in_image, in_image, image_size, hipMemcpyHostToDevice);

    // Kernel launch
    dim3 grid_image(width, height);
    motion_blur_cuda << <grid_image, 1 >> > (device_in_image, device_out_image, kernel_size, channels, height, width, angle_deg, distance);

    // Copy the device variables to the host (GPU -> CPU)
    hipMemcpy(out_image, device_out_image, image_size, hipMemcpyDeviceToHost);

    // Free up the memory on the device (GPU)
    hipFree(device_in_image);
    hipFree(device_out_image);
}

__global__ void motion_blur_cuda(
    const unsigned char* in_image,
    unsigned char* out_image,
    const int kernel_size,
    const int channels,
    const int height,
    const int width,
    const float angle_deg,
    const int distance
)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = (x + y * width) * channels;

    if (x < width && y < height)
    {
        int start_kernel_x = x - kernel_size / 2;
        int start_kernel_y = y - kernel_size / 2;

        for (int channel = 0; channel < channels; channel++)
        {
            double channel_sum = 0;
            for (int x_kernel = start_kernel_x; x_kernel < start_kernel_x + kernel_size; x_kernel++)
            {
                if (x_kernel < 0 || x_kernel >= width)
                {
                    continue;
                }

                for (int y_kernel = start_kernel_y; y_kernel < start_kernel_y + kernel_size; y_kernel++)
                {
                    if (y_kernel < 0 || y_kernel >= height)
                    {
                        continue;
                    }

                    unsigned char pixel = in_image[(x_kernel + y_kernel * width) * channels + channel];

                    int kernel_index = (x_kernel - start_kernel_x) + (y_kernel - start_kernel_y) * kernel_size;
                    unsigned char kernel_value = testBit(device_kernel, kernel_index);

                    if (kernel_value)
                    {
                        channel_sum += static_cast<double>(pixel) / (distance + 1);
                    }
                }
            }
            out_image[index + channel] = static_cast<unsigned char>(channel_sum);
        }
    }
}
